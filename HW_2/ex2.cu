/* compile with: nvcc -O3 -maxrregcount=32 ex2.cu -o ex2 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <time.h>
#include <assert.h>
#include <string.h>

///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
#define IMG_DIMENSION 32
#define NREQUESTS 10000
#define N_STREAMS 64
#define INVALID -1
#define VALID 1
#define Q_SLOTS 10
#define MAX_REGISTER_COUNT 32
#define SQR(a) ((a) * (a))
#define PIXEL_VALUES 256
#define SHARED_MEM_PER_BLOCK 3 * PIXEL_VALUES

#define INCREASE_PC_POINTER(X) ((X) + 1) % Q_SLOTS

typedef unsigned char uchar;

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

bool is_queue_full(int *p_id, int *c_id) {
    __sync_synchronize();
    return INCREASE_PC_POINTER(*p_id) == *c_id;
}

void enqueueJob(uchar *queue, int *p_id, int image_idx,uchar * images_in, int image_size, uchar valid){
    if(valid == VALID){
        memcpy(queue + (((*p_id) * (1 + image_size)) + 1), images_in + (image_idx * image_size), image_size);
    }
    memcpy(queue + (*p_id * ( 1 + image_size)), &valid, sizeof(uchar));
    __sync_synchronize();
    *p_id = INCREASE_PC_POINTER(*p_id);
    __sync_synchronize();
}

void dequeueJob(uchar *queue, int c_id, int image_idx, uchar *images_out, int image_size){
    memcpy(images_out + (image_idx * image_size), queue + (c_id * image_size), image_size);
    __sync_synchronize();
}

int numOfThreadBlocksCalc(int threadsPerBlock) {
    int sharedMemPerBlock = SHARED_MEM_PER_BLOCK;
    int regsPerBlock = threadsPerBlock * MAX_REGISTER_COUNT;

    hipDeviceProp_t currDeviceProperties;

    CUDA_CHECK(hipGetDeviceProperties(&currDeviceProperties, 0));
    // hardware limitation
    int numOfBlocksPerSMSharedMem = currDeviceProperties.sharedMemPerMultiprocessor / sharedMemPerBlock;
    int numOfBlocksPerSMRegs = currDeviceProperties.regsPerMultiprocessor / regsPerBlock;
    int numOfBlocksPerSMThreads = currDeviceProperties.maxThreadsPerMultiProcessor / threadsPerBlock;

    //Get the minimum threadBlock amount per multiProcessor subject to hardware limitation
    int minBlocksPerSM = numOfBlocksPerSMSharedMem;
    if (numOfBlocksPerSMRegs < minBlocksPerSM) minBlocksPerSM = numOfBlocksPerSMRegs;
    if (numOfBlocksPerSMThreads < minBlocksPerSM) minBlocksPerSM = numOfBlocksPerSMThreads;

    //the threadBlock amount is per SM multiply by number of SMs
    return minBlocksPerSM * currDeviceProperties.multiProcessorCount;
}

bool is_empty(int *c_id, int *p_id) {
    __sync_synchronize();
    return *c_id == *p_id;
}

void process_image(uchar *img_in, uchar *img_out) {
    int histogram[256] = { 0 };
    for (int i = 0; i < SQR(IMG_DIMENSION); i++) {
        histogram[img_in[i]]++;
    }

    int cdf[256] = { 0 };
    int hist_sum = 0;
    for (int i = 0; i < 256; i++) {
        hist_sum += histogram[i];
        cdf[i] = hist_sum;
    }

    int cdf_min = 0;
    for (int i = 0; i < 256; i++) {
        if (cdf[i] != 0) {
            cdf_min = cdf[i];
            break;
        }
    }

    uchar map[256] = { 0 };
    for (int i = 0; i < 256; i++) {
        int map_value = (float)(cdf[i] - cdf_min) / (SQR(IMG_DIMENSION) - cdf_min) * 255;
        map[i] = (uchar)map_value;
    }

    for (int i = 0; i < SQR(IMG_DIMENSION); i++) {
        img_out[i] = map[img_in[i]];
    }
}

double static inline get_time_msec(void) {
    struct timespec t;
    int res = clock_gettime(CLOCK_MONOTONIC, &t);
    if (res) {
        perror("clock_gettime failed");
        exit(1);
    }
    return t.tv_sec * 1e+3 + t.tv_nsec * 1e-6;
}

/* we'll use these to rate limit the request load */
struct rate_limit_t {
    double last_checked;
    double lambda;
    unsigned seed;
};

void rate_limit_init(struct rate_limit_t *rate_limit, double lambda, int seed) {
    rate_limit->lambda = lambda;
    rate_limit->seed = (seed == -1) ? 0 : seed;
    rate_limit->last_checked = 0;
}

int rate_limit_can_send(struct rate_limit_t *rate_limit) {
    if (rate_limit->lambda == 0) return 1;
    double now = get_time_msec() * 1e-3;
    double dt = now - rate_limit->last_checked;
    double p = dt * rate_limit->lambda;
    rate_limit->last_checked = now;
    if (p > 1) p = 1;
    double r = (double)rand_r(&rate_limit->seed) / RAND_MAX;
    return (p > r);
}

void rate_limit_wait(struct rate_limit_t *rate_limit) {
    while (!rate_limit_can_send(rate_limit)) {
        struct timespec t = {
            0,
            long(1. / (rate_limit->lambda * 1e-9) * 0.01)
        };
        nanosleep(&t, NULL);
    }
}

double distance_sqr_between_image_arrays(uchar *img_arr1, uchar *img_arr2) {
    double distance_sqr = 0;
    for (int i = 0; i < NREQUESTS * SQR(IMG_DIMENSION); i++) {
        distance_sqr += SQR(img_arr1[i] - img_arr2[i]);
    }
    return distance_sqr;
}

/* we won't load actual files. just fill the images with random bytes */
void load_images(uchar *images) {
    srand(0);
    for (int i = 0; i < NREQUESTS * SQR(IMG_DIMENSION); i++) {
        images[i] = rand() % 256;
    }
}

__device__ int arr_min(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int rhs, lhs;

    for (int stride = 1; stride < arr_size; stride *= 2) {
        if (tid >= stride && tid < arr_size) {
            rhs = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && tid < arr_size) {
            lhs = arr[tid];
            if (rhs != 0) {
                if (lhs == 0)
                    arr[tid] = rhs;
                else
                    arr[tid] = min(arr[tid], rhs);
            }
        }
        __syncthreads();
    }

    int ret = arr[arr_size - 1];
    return ret;
}

__device__ void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;

    for (int stride = 1; stride < min(blockDim.x, arr_size); stride *= 2) {
        if (tid >= stride && tid < arr_size) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && tid < arr_size) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
}

__global__ void gpu_process_image(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ int hist_min[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        hist_min[tid] = histogram[tid];
    }
    __syncthreads();

    int cdf_min = arr_min(hist_min, 256);

    __shared__ uchar map[256];
    if (tid < 256) {
        int map_value = (float)(histogram[tid] - cdf_min) / (SQR(IMG_DIMENSION) - cdf_min) * 255;
        map[tid] = (uchar)map_value;
    }

    __syncthreads();

    for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x) {
        out[i] = map[in[i]];
    }
    return;
}

__global__ void gpu_server(int* cpu2gpu_producer_idx_g, int* cpu2gpu_consumer_idx_g,uchar* cpu_gpu_queue_g, uchar* gpu_cpu_queue_g){
    __shared__ int histogram[PIXEL_VALUES];
    __shared__ int hist_min[PIXEL_VALUES];
    __shared__ uchar map[PIXEL_VALUES];
    __shared__ volatile uchar req;
    volatile uchar *image_in;
    int slot_size_bytes = 1 + SQR(IMG_DIMENSION);
    int tid = threadIdx.x;
    while (1) {
        if (tid == 0) {
            /* busy wait while there are no outstanding jobs or gpu_cpu_queue is full */
            int current_consume_slot = cpu2gpu_consumer_idx_g[blockIdx.x];
            while (cpu2gpu_producer_idx_g[blockIdx.x] == current_consume_slot){
                __threadfence_system();
            }
            uchar * tb_cpu_gpu_queue = cpu_gpu_queue_g + (blockIdx.x * Q_SLOTS  * slot_size_bytes);
            __threadfence_block();
            image_in = tb_cpu_gpu_queue + (current_consume_slot * slot_size_bytes);
            __threadfence_block();
            req = image_in[0];//check validator byte
            __threadfence_block();
            image_in++;//remove validator byte
            __threadfence_block();
        }
        __syncthreads();
        if (req != VALID){
            return;
        }

        if (tid < PIXEL_VALUES) {
            histogram[tid] = 0;
        }
        __threadfence_block();
        int current_consume_slot = cpu2gpu_consumer_idx_g[blockIdx.x];
        uchar * tb_cpu_gpu_queue = cpu_gpu_queue_g + (blockIdx.x * Q_SLOTS  * slot_size_bytes);
        image_in = tb_cpu_gpu_queue + (current_consume_slot * slot_size_bytes) + 1;

//        for (int i = tid; i < SQR(IMG_DIMENSION); i += gr.x) {
            atomicAdd(&histogram[image_in[tid]], 1);
//        }
        __threadfence_block();

        prefix_sum(histogram, PIXEL_VALUES);
        __threadfence_block();

        if (tid < PIXEL_VALUES) {
            hist_min[tid] = histogram[tid];
        }
        __threadfence_block();

        int cdf_min = arr_min(hist_min, PIXEL_VALUES);
        __threadfence_block();

        if (tid < PIXEL_VALUES) {
            int map_value = (float)(histogram[tid] - cdf_min) / (SQR(IMG_DIMENSION) - cdf_min) * 255;
            map[tid] = (uchar)map_value;
        }

        __threadfence_block();
        uchar * tb_gpu_cpu_queue = gpu_cpu_queue_g + (blockIdx.x * Q_SLOTS  * SQR(IMG_DIMENSION));
        uchar *current_start_slot = tb_gpu_cpu_queue + (SQR(IMG_DIMENSION) * current_consume_slot);
        __threadfence_block();

//        for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x) {
            current_start_slot[tid] =  map[image_in[tid]];
//        }
        __threadfence_system();

        if (tid == 0) {
            cpu2gpu_consumer_idx_g[blockIdx.x] = INCREASE_PC_POINTER(cpu2gpu_consumer_idx_g[blockIdx.x]);
            __threadfence_system();
        }
        __syncthreads();
    }
}

void print_usage_and_die(char *progname) {
    printf("usage:\n");
    printf("%s streams <load (requests/sec)>\n", progname);
    printf("OR\n");
    printf("%s queue <#threads> <load (requests/sec)>\n", progname);
    exit(1);
}


enum {PROGRAM_MODE_STREAMS = 0, PROGRAM_MODE_QUEUE};
int main(int argc, char *argv[]) {

    int mode = -1;
    int threads_queue_mode = -1; /* valid only when mode = queue */
    double load = 0;
    if (argc < 3) print_usage_and_die(argv[0]);

    if (!strcmp(argv[1], "streams")) {
        if (argc != 3) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_STREAMS;
        load = atof(argv[2]);
    } else if (!strcmp(argv[1], "queue")) {
        if (argc != 4) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_QUEUE;
        threads_queue_mode = atoi(argv[2]);
        load = atof(argv[3]);
    } else {
        print_usage_and_die(argv[0]);
    }

    uchar *images_in; /* we concatenate all images in one huge array */
    uchar *images_out;
    CUDA_CHECK( hipHostAlloc(&images_in, NREQUESTS * SQR(IMG_DIMENSION), 0) );
    CUDA_CHECK( hipHostAlloc(&images_out, NREQUESTS * SQR(IMG_DIMENSION), 0) );

    load_images(images_in);
    double t_start, t_finish;

    /* using CPU */
    printf("\n=== CPU ===\n");
    t_start  = get_time_msec();
    for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx)
        process_image(&images_in[img_idx * SQR(IMG_DIMENSION)], &images_out[img_idx * SQR(IMG_DIMENSION)]);
    t_finish = get_time_msec();
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

    double total_distance = 0;

    /* using GPU task-serial.. just to verify the GPU code makes sense */
    printf("\n=== GPU Task Serial ===\n");

    uchar *images_out_from_gpu;
    CUDA_CHECK( hipHostAlloc(&images_out_from_gpu, NREQUESTS * SQR(IMG_DIMENSION), 0) );

    do {
        uchar *gpu_image_in, *gpu_image_out;
        CUDA_CHECK(hipMalloc(&gpu_image_in, SQR(IMG_DIMENSION)));
        CUDA_CHECK(hipMalloc(&gpu_image_out, SQR(IMG_DIMENSION)));

        t_start = get_time_msec();
        for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx) {
            CUDA_CHECK(hipMemcpy(gpu_image_in, &images_in[img_idx * SQR(IMG_DIMENSION)], SQR(IMG_DIMENSION), hipMemcpyHostToDevice));
            gpu_process_image<<<1, 1024>>>(gpu_image_in, gpu_image_out);
            CUDA_CHECK(hipMemcpy(&images_out_from_gpu[img_idx * SQR(IMG_DIMENSION)], gpu_image_out, SQR(IMG_DIMENSION), hipMemcpyDeviceToHost));
        }
        total_distance += distance_sqr_between_image_arrays(images_out, images_out_from_gpu);
        CUDA_CHECK(hipDeviceSynchronize());
        t_finish = get_time_msec();
        printf("distance from baseline %lf (should be zero)\n", total_distance);
        printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

        CUDA_CHECK(hipFree(gpu_image_in));
        CUDA_CHECK(hipFree(gpu_image_out));
    } while (0);

    /* now for the client-server part */
    printf("\n=== Client-Server ===\n");
    double *req_t_start = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_start, 0, NREQUESTS * sizeof(double));

    double *req_t_end = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_end, 0, NREQUESTS * sizeof(double));

    struct rate_limit_t rate_limit;
    rate_limit_init(&rate_limit, load, 0);

    CUDA_CHECK(hipMemset(images_out_from_gpu, 0, NREQUESTS * SQR(IMG_DIMENSION)));

    const int IMAGE_SIZE = SQR(IMG_DIMENSION);

    double ti = get_time_msec();
    if (mode == PROGRAM_MODE_STREAMS) {
        //declare streams in and out buffers
        uchar *image_in_device_streams, *image_out_device_streams;
        /* allocating device memory for all number of streams * image size */
        CUDA_CHECK(hipMalloc((void **)&image_in_device_streams, N_STREAMS * SQR(IMG_DIMENSION)));
        CUDA_CHECK(hipMalloc((void **)&image_out_device_streams,N_STREAMS * SQR(IMG_DIMENSION)));

        /* initialize CUDA streams*/
        hipStream_t streams[N_STREAMS];
        for (int i = 0; i < N_STREAMS; i++) {
            CUDA_CHECK(hipStreamCreate(&streams[i]));
        }

        /* save per streams current handle request*/
        int request_per_stream[N_STREAMS];
        for(int s_i=0; s_i < N_STREAMS; s_i++){
            request_per_stream[s_i] = INVALID;
        }

        for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx) {
            int chosen_stream = INVALID;
            /*finding stream to perform request */
            while (chosen_stream == INVALID){
                /* freeing stresms who finished */
                for (int stream_i = 0; stream_i < N_STREAMS; stream_i ++) {
                    if (request_per_stream[stream_i] != INVALID){// true when stream perform a request
                        if (hipStreamQuery(streams[stream_i]) != hipSuccess){//check if stream finished the job
                            continue;
                        }
                        req_t_end[request_per_stream[stream_i]] = get_time_msec();//record finished time
                        request_per_stream[stream_i] = INVALID;//mark stream job free
                    }
                    if (chosen_stream == INVALID){
                        chosen_stream = stream_i;//take stream to do next job
                    }
                }
            }
            if (!rate_limit_can_send(&rate_limit)) {
                --img_idx;
                continue;
            }
            req_t_start[img_idx] = get_time_msec();
            request_per_stream[chosen_stream] = img_idx;// recored stream current job

            CUDA_CHECK(hipMemcpyAsync(image_in_device_streams + (chosen_stream * IMAGE_SIZE),
                                       images_in + (img_idx * IMAGE_SIZE), IMAGE_SIZE,
                                       hipMemcpyHostToDevice, streams[chosen_stream]));
            gpu_process_image <<< 1, 1024, 0, streams[chosen_stream] >>> (image_in_device_streams + (chosen_stream * IMAGE_SIZE), image_out_device_streams + (chosen_stream * IMAGE_SIZE));
            CUDA_CHECK(hipMemcpyAsync(images_out_from_gpu + (img_idx * IMAGE_SIZE),
                                       image_out_device_streams + (chosen_stream * IMAGE_SIZE),
                                       IMAGE_SIZE, hipMemcpyDeviceToHost, streams[chosen_stream]));
        }

        CUDA_CHECK(hipDeviceSynchronize());//wait all job to finished
        bool all_done = false;
        while (!all_done){
            all_done = true;
            /* freeing streams who finished */
            for (int stream_i = 0; stream_i < N_STREAMS; stream_i ++) {
                if (request_per_stream[stream_i] != INVALID){// true when stream perform a request
                    if (hipStreamQuery(streams[stream_i]) != hipSuccess){//check if stream finished the job
                        all_done = false;
                        continue;
                    }
                    req_t_end[request_per_stream[stream_i]] = get_time_msec();//record finished time
                    request_per_stream[stream_i] = INVALID;//mark stream job free
                }
            }
        }
        // cleanup streams environment
        for (int i = 0; i < N_STREAMS; i++) {
            CUDA_CHECK(hipStreamDestroy(streams[i]));
        }
        CUDA_CHECK(hipFree(image_out_device_streams));
        CUDA_CHECK(hipFree(image_in_device_streams));
    }else if (mode == PROGRAM_MODE_QUEUE) {
//        int numberOfThreadBlocks = numOfThreadBlocksCalc(threads_queue_mode);
//         printf("Number of ThreadBlocks is %d\n", numberOfThreadBlocks);
//
//        // memory alloc
//        uchar *cpu_gpu_queue, *cpu_gpu_queue_g,*gpu_cpu_queue, *gpu_cpu_queue_g;
//
//        CUDA_CHECK(cudaHostAlloc(&cpu_gpu_queue, numberOfThreadBlocks * Q_SLOTS * (1 + image_size), 0));
//        CUDA_CHECK(cudaHostAlloc(&gpu_cpu_queue, numberOfThreadBlocks * Q_SLOTS * image_size, 0));
//
//        int *cpu2gpu_producer_idx, *cpu2gpu_consumer_idx;
//        int *cpu2gpu_producer_idx_g, *cpu2gpu_consumer_idx_g;
//
//        CUDA_CHECK(cudaHostAlloc(&cpu2gpu_producer_idx, numberOfThreadBlocks, 0));
//        CUDA_CHECK(cudaHostAlloc(&cpu2gpu_consumer_idx, numberOfThreadBlocks, 0));
//
//        CUDA_CHECK(cudaHostGetDevicePointer(&cpu2gpu_producer_idx_g, cpu2gpu_producer_idx, 0));
//        CUDA_CHECK(cudaHostGetDevicePointer(&cpu2gpu_consumer_idx_g, cpu2gpu_consumer_idx, 0));
//        CUDA_CHECK(cudaHostGetDevicePointer(&cpu_gpu_queue_g, cpu_gpu_queue, 0));
//        CUDA_CHECK(cudaHostGetDevicePointer(&gpu_cpu_queue_g, gpu_cpu_queue, 0));
//
//        int *req_by_tb = (int *) malloc(numberOfThreadBlocks * Q_SLOTS * sizeof(int));
//        int *last_slot_by_tb= (int *) malloc(numberOfThreadBlocks * sizeof(int));
//
//        //memsets
//        memset(cpu2gpu_producer_idx, 0, numberOfThreadBlocks * sizeof(int));
//        memset(cpu2gpu_consumer_idx, 0, numberOfThreadBlocks * sizeof(int));
//        memset(cpu_gpu_queue, 0, numberOfThreadBlocks * Q_SLOTS * (1 + image_size));
//        memset(gpu_cpu_queue, 0,  numberOfThreadBlocks * Q_SLOTS * image_size);
//        memset(last_slot_by_tb, 0, numberOfThreadBlocks * sizeof(int));
//        memset(req_by_tb, INVALID, numberOfThreadBlocks * Q_SLOTS * sizeof(int));
//
//        gpu_server <<< numberOfThreadBlocks, threads_queue_mode >>>
//                                              (cpu2gpu_producer_idx_g, cpu2gpu_consumer_idx_g, cpu_gpu_queue_g, gpu_cpu_queue_g);
//
//        for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx) {
//            printf("\ncurrent request: %d\n", img_idx);
//            int chosenThreadBlock = INVALID;
//            while (chosenThreadBlock == INVALID) {
//                for (int tb_it = 0; tb_it < numberOfThreadBlocks; tb_it++) {
//                    // read completed requests from tb
//                    while (!is_empty(&last_slot_by_tb[tb_it], &cpu2gpu_consumer_idx[tb_it])) {
//                        printf("tb %d queue is not empty\n", tb_it);
//                        int completeRequest = req_by_tb[(tb_it * Q_SLOTS) + last_slot_by_tb[tb_it]];
//                        printf("ThreadBlock id %d, request handled is %d, last_slot is %d, p_idx is %d, c_idx is %d\n",
//                               tb_it, completeRequest, last_slot_by_tb[tb_it], cpu2gpu_producer_idx[tb_it],
//                               cpu2gpu_consumer_idx[tb_it]);
//                        req_t_end[completeRequest] = get_time_msec();
//                        req_by_tb[(tb_it * Q_SLOTS) + last_slot_by_tb[tb_it]] = INVALID;
//                        dequeueJob(gpu_cpu_queue + (tb_it * Q_SLOTS), last_slot_by_tb[tb_it], completeRequest,
//                                   images_out_from_gpu, image_size);
//                        last_slot_by_tb[tb_it] = INCREASE_PC_POINTER(last_slot_by_tb[tb_it]);
//                    }
//                    if (chosenThreadBlock == INVALID &&
//                        !is_queue_full(cpu2gpu_producer_idx + tb_it, cpu2gpu_consumer_idx + tb_it)) {
//                        chosenThreadBlock = tb_it;
//                    }
//                }
//            }
//            if (!rate_limit_can_send(&rate_limit)) {
//                --img_idx;
//                continue;
//            }
//            req_t_start[img_idx] = get_time_msec();
//            req_by_tb[chosenThreadBlock * Q_SLOTS + cpu2gpu_producer_idx[chosenThreadBlock]] = img_idx;
//            printf("enqueue job %d to threadBlock %d p_id %d\n",img_idx, chosenThreadBlock,cpu2gpu_producer_idx[chosenThreadBlock]);
//            enqueueJob(cpu_gpu_queue + (chosenThreadBlock * Q_SLOTS * ((1 + image_size))), &cpu2gpu_producer_idx[chosenThreadBlock],
//                       img_idx, images_in, image_size, VALID);
//        }
//            /* wait until you have responses for all requests */
//            for (int tb_it = 0; tb_it < numberOfThreadBlocks; tb_it++) {
//                while (is_queue_full(cpu2gpu_producer_idx + tb_it, cpu2gpu_consumer_idx + tb_it)) {
//                }
//                req_by_tb[(tb_it * Q_SLOTS) + cpu2gpu_producer_idx[tb_it]] = INVALID;
//                enqueueJob(cpu_gpu_queue + (tb_it * Q_SLOTS * ((1 + image_size))),
//                        &cpu2gpu_producer_idx[tb_it],
//                        0, images_in, image_size, 0);
//                printf("added not valid request 0 to tb %d, p id %d c id %d l id %d\n", tb_it,
//                       cpu2gpu_producer_idx[tb_it], cpu2gpu_consumer_idx[tb_it], last_slot_by_tb[tb_it]);
//            }
//            CUDA_CHECK(cudaDeviceSynchronize());
//            printf("all devices finished!\n");
//            for (int tb_it = 0; tb_it < numberOfThreadBlocks; tb_it++) {
//                // read completed requests from tb
//                while (!is_empty(&last_slot_by_tb[tb_it], &cpu2gpu_consumer_idx[tb_it])) {
//                    int completeRequest = req_by_tb[(tb_it * Q_SLOTS) + last_slot_by_tb[tb_it]];
//                    printf("ThreadBlock id %d, request handled is %d, last_slot is %d, p_idx is %d, c_idx is %d\n",
//                           tb_it, completeRequest, last_slot_by_tb[tb_it], cpu2gpu_producer_idx[tb_it],
//                           cpu2gpu_consumer_idx[tb_it]);
//                    if (completeRequest != -1) {
//                        req_t_end[completeRequest] = get_time_msec();
//                        dequeueJob(gpu_cpu_queue + (tb_it * Q_SLOTS), last_slot_by_tb[tb_it], completeRequest,
//                                   images_out_from_gpu, image_size);
//                    }
//                    last_slot_by_tb[tb_it] = INCREASE_PC_POINTER(last_slot_by_tb[tb_it]);
//                }
//            }
        }
    else {
        assert(0);
    }
    double tf = get_time_msec();

    total_distance = distance_sqr_between_image_arrays(images_out, images_out_from_gpu);
    double avg_latency = 0;
    for (int i = 0; i < NREQUESTS; i++) {
        avg_latency += (req_t_end[i] - req_t_start[i]);
    }
    avg_latency /= NREQUESTS;

    printf("mode = %s\n", mode == PROGRAM_MODE_STREAMS ? "streams" : "queue");
    printf("load = %lf (req/sec)\n", load);
    if (mode == PROGRAM_MODE_QUEUE) printf("threads = %d\n", threads_queue_mode);
    printf("distance from baseline %lf (should be zero)\n", total_distance);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (tf - ti) * 1e+3);
    printf("average latency = %lf (msec)\n", avg_latency);
    return 0;
}
