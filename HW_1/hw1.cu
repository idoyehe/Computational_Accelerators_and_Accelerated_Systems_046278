/* compile with: nvcc -O3 hw1.cu -o hw1 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
#define IMG_HEIGHT 256
#define IMG_WIDTH 256
#define N_IMAGES 10000
#define HISTOGRAM_SIZE 256

typedef unsigned char uchar;

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

void process_image(uchar *img_in, uchar *img_out) {
    int histogram[256] = { 0 };
    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        histogram[img_in[i]]++;
    }

    int cdf[256] = { 0 };
    int hist_sum = 0;
    for (int i = 0; i < 256; i++) {
        hist_sum += histogram[i];
        cdf[i] = hist_sum;
    }

    int cdf_min = 0;
    for (int i = 0; i < 256; i++) {
        if (cdf[i] != 0) {
            cdf_min = cdf[i];
            break;
        }
    }

    uchar map[256] = { 0 };
    for (int i = 0; i < 256; i++) {
        int map_value = (float)(cdf[i] - cdf_min) / (IMG_WIDTH * IMG_HEIGHT - cdf_min) * 255;
        map[i] = (uchar)map_value;
    }

    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        img_out[i] = map[img_in[i]];
    }
}

double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}

long long int distance_sqr_between_image_arrays(uchar *img_arr1, uchar *img_arr2) {
    long long int distance_sqr = 0;
    for (int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
        distance_sqr += SQR(img_arr1[i] - img_arr2[i]);
    }
    return distance_sqr;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int array_min_positive(int *arr, int len){
    int tid = threadIdx.x;
    __shared__ int min_arr[HISTOGRAM_SIZE];
    if (tid < len) {
        min_arr[tid] = arr[tid]; //copy the arr to preserve it
    }
    __syncthreads();
    int half_size = len /2;
    while (half_size >=1){
        if (tid < half_size) {
            bool change_flag = (min_arr[tid + half_size] > 0 && min_arr[tid]
                    > min_arr[tid + half_size] || min_arr[tid] == 0);
            min_arr[tid] = change_flag * min_arr[tid + half_size] +
                           (!change_flag) * min_arr[tid];
        }
        __syncthreads();
        half_size /=2;
    }
    return min_arr[0];
}

__device__ void prefix_sum(int *arr, int len){
    int tid = threadIdx.x;
    int increment;
    for (int stride = 1; stride < len; stride *= 2) {
        if (tid < len && tid >= stride) { // in case # threads bigger than array length
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid < len && tid >= stride) { // in case # threads bigger than array length
            arr[tid] += increment;
        }
        __syncthreads();
    }
    return;
}

__device__ void map(int *cdf, int cdfMin, uchar* mapOut, int len){
    int tid = threadIdx.x;
    if (tid < len) {
        int map_value = (float)(cdf[tid] - cdfMin) / (IMG_WIDTH * IMG_HEIGHT - cdfMin) * 255;
        mapOut[tid] =(uchar)map_value;
    }
    return;
}

__global__ void process_image_kernel(uchar *in, uchar *out) {
    int tid = threadIdx.x;
    int imageStartIndex = IMG_WIDTH * IMG_HEIGHT * blockIdx.x;
    __shared__ int hist_shared[HISTOGRAM_SIZE];
    if (tid < HISTOGRAM_SIZE) {
        hist_shared[tid] = 0;
    }

    for(int startOffset = 0; startOffset < IMG_WIDTH * IMG_HEIGHT; startOffset += blockDim.x){
        int pixelValue = in[imageStartIndex + startOffset + tid];
        atomicAdd(hist_shared + pixelValue, 1);
    }
    __syncthreads();
    prefix_sum(hist_shared, HISTOGRAM_SIZE);
    int * cdf = hist_shared;
    __syncthreads();
    int cdfMin = array_min_positive(cdf, HISTOGRAM_SIZE);
    __syncthreads();
    __shared__ uchar mapOut[HISTOGRAM_SIZE];
    map(cdf, cdfMin, mapOut, HISTOGRAM_SIZE);
    __syncthreads();
    for(int startOffset = 0; startOffset < IMG_WIDTH * IMG_HEIGHT; startOffset += blockDim.x){
        int pixelValue = in[imageStartIndex + startOffset + tid];
        out[imageStartIndex + startOffset + tid] = mapOut[pixelValue];
    }
    return;
}

int main() {
///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
    uchar *images_in;
    uchar *images_out_cpu; //output of CPU computation. In CPU memory.
    uchar *images_out_gpu_serial; //output of GPU task serial computation. In CPU memory.
    uchar *images_out_gpu_bulk; //output of GPU bulk computation. In CPU memory.
    CUDA_CHECK( hipHostAlloc(&images_in, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_cpu, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_serial, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_bulk, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );

    /* instead of loading real images, we'll load the arrays with random data */
    srand(0);
    for (long long int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
        images_in[i] = rand() % 256;
    }

    double t_start, t_finish;

    // CPU computation. For reference. Do not change
    printf("\n=== CPU ===\n");
    t_start = get_time_msec();
    for (int i = 0; i < N_IMAGES; i++) {
        uchar *img_in = &images_in[i * IMG_WIDTH * IMG_HEIGHT];
        uchar *img_out = &images_out_cpu[i * IMG_WIDTH * IMG_HEIGHT];
        process_image(img_in, img_out);
    }
    t_finish = get_time_msec();
    printf("total time %f [msec]\n", t_finish - t_start);

    long long int distance_sqr;
///////////////////////////////////////////////////////////////////////////////////////////////////////////

    // GPU task serial computation
    printf("\n=== GPU Task Serial ===\n"); //Do not change
    uchar *image_in_device_serial, *image_out_device_serial;
    CUDA_CHECK(hipMalloc((void **)&image_in_device_serial,IMG_HEIGHT * IMG_WIDTH ));
    CUDA_CHECK(hipMalloc((void **)&image_out_device_serial,IMG_HEIGHT * IMG_WIDTH ));
    t_start = get_time_msec(); //Do not change
    for (int i = 0; i < N_IMAGES; i++) {
        int imageStartIndex =  IMG_HEIGHT * IMG_WIDTH * i;
        CUDA_CHECK(hipMemcpy(image_in_device_serial, images_in + imageStartIndex,
                              IMG_HEIGHT * IMG_WIDTH,
                              hipMemcpyHostToDevice));
        process_image_kernel <<< 1, 1024 >>> (image_in_device_serial, image_out_device_serial);

        CUDA_CHECK(hipMemcpy(images_out_gpu_serial + imageStartIndex, image_out_device_serial,
                              IMG_HEIGHT * IMG_WIDTH, hipMemcpyDeviceToHost));
    }
    t_finish = get_time_msec(); //Do not change
    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_serial); // Do not change
    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not change

    // GPU bulk
    printf("\n=== GPU Bulk ===\n"); //Do not change
    uchar *image_in_device_bulk, *image_out_device_bulk;
    CUDA_CHECK(hipMalloc((void **)&image_in_device_bulk,IMG_HEIGHT * IMG_WIDTH * N_IMAGES ));
    CUDA_CHECK(hipMalloc((void **)&image_out_device_bulk,IMG_HEIGHT * IMG_WIDTH * N_IMAGES ));
    t_start = get_time_msec(); //Do not change
    CUDA_CHECK(hipMemcpy(image_in_device_bulk, images_in, IMG_HEIGHT * IMG_WIDTH * N_IMAGES, hipMemcpyHostToDevice));
    process_image_kernel <<< N_IMAGES, 1024 >>> (image_in_device_bulk, image_out_device_bulk);
    CUDA_CHECK(hipMemcpy(images_out_gpu_bulk, image_out_device_bulk, IMG_HEIGHT * IMG_WIDTH * N_IMAGES, hipMemcpyDeviceToHost));
    t_finish = get_time_msec(); //Do not change
    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_bulk); // Do not change
    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not change
    return 0;
}