/* compile with: nvcc -O3 hw1.cu -o hw1 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
#define IMG_HEIGHT 256
#define IMG_WIDTH 256
#define N_IMAGES 10000
#define HISTOGRAM_SIZE 256

typedef unsigned char uchar;

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

void process_image(uchar *img_in, uchar *img_out) {
    int histogram[256] = { 0 };
    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        histogram[img_in[i]]++;
    }

    int cdf[256] = { 0 };
    int hist_sum = 0;
    for (int i = 0; i < 256; i++) {
        hist_sum += histogram[i];
        cdf[i] = hist_sum;
    }

    int cdf_min = 0;
    for (int i = 0; i < 256; i++) {
        if (cdf[i] != 0) {
            cdf_min = cdf[i];
            break;
        }
    }

    uchar map[256] = { 0 };
    for (int i = 0; i < 256; i++) {
        int map_value = (float)(cdf[i] - cdf_min) / (IMG_WIDTH * IMG_HEIGHT - cdf_min) * 255;
        map[i] = (uchar)map_value;
    }

    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        img_out[i] = map[img_in[i]];
    }
}

double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}

long long int distance_sqr_between_image_arrays(uchar *img_arr1, uchar *img_arr2) {
    long long int distance_sqr = 0;
    for (int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
        distance_sqr += SQR(img_arr1[i] - img_arr2[i]);
    }
    return distance_sqr;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int array_min_positive(int *arr, int len){
    int tid = threadIdx.x;
    __shared__ uchar min_arr[HISTOGRAM_SIZE];
    min_arr[tid] = arr[tid]; //copy the arr to preserve it
    int half_size = len /2;
    while (half_size >=1){
        if (tid < half_size) {
            bool change_flag = (min_arr[tid + half_size] > 0 && min_arr[tid]
                    > min_arr[tid + half_size] || min_arr[tid] == 0);
            min_arr[tid] = change_flag * min_arr[tid + half_size] +
                           (!change_flag) * min_arr[tid];
        }
        __syncthreads();
        half_size /=2;
    }
    return min_arr[0];
}

__device__ void prefix_sum(int *arr, int len){
    int tid = threadIdx.x;
    int increment;
    for (int stride = 1; stride < len; stride *= 2) {
        if (tid < len && tid >= stride) { // in case # threads bigger than array length
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid < len && tid >= stride) { // in case # threads bigger than array length
            arr[tid] += increment;
        }
        __syncthreads();
    }
    return;
}

__global__ void process_image_kernel(int *in, int *out) {
    int tid = threadIdx.x;
    int res = array_min_positive(in,HISTOGRAM_SIZE);
    prefix_sum(in,HISTOGRAM_SIZE);
    out[tid]=in[tid];
    if(tid ==0) {
        out[HISTOGRAM_SIZE] = res;
    }
    return ;
}

int main() {
///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
    uchar *images_in;
    uchar *images_out_cpu; //output of CPU computation. In CPU memory.
    uchar *images_out_gpu_serial; //output of GPU task serial computation. In CPU memory.
    uchar *images_out_gpu_bulk; //output of GPU bulk computation. In CPU memory.
    CUDA_CHECK( hipHostAlloc(&images_in, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_cpu, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_serial, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_bulk, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );

//    /* instead of loading real images, we'll load the arrays with random data */
//    srand(0);
//    for (long long int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
//        images_in[i] = rand() % 256;
//    }
//
//    double t_start, t_finish;
//
//    // CPU computation. For reference. Do not change
//    printf("\n=== CPU ===\n");
//    t_start = get_time_msec();
//    for (int i = 0; i < N_IMAGES; i++) {
//        uchar *img_in = &images_in[i * IMG_WIDTH * IMG_HEIGHT];
//        uchar *img_out = &images_out_cpu[i * IMG_WIDTH * IMG_HEIGHT];
//        process_image(img_in, img_out);
//    }
//    t_finish = get_time_msec();
//    printf("total time %f [msec]\n", t_finish - t_start);
//
//    long long int distance_sqr;
///////////////////////////////////////////////////////////////////////////////////////////////////////////

    // GPU task serial computation
    printf("\n=== GPU Task Serial ===\n"); //Do not change
    int *image_in_device_serial, *image_out_device_serial;
    CUDA_CHECK(hipMalloc((void **)&image_in_device_serial,HISTOGRAM_SIZE * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&image_out_device_serial,(1+HISTOGRAM_SIZE) * sizeof(int)));

    //TODO: allocate GPU memory for a single input image and a single output image
//    t_start = get_time_msec(); //Do not change
    int* temp;
    temp =(int*)malloc(sizeof(int)*(HISTOGRAM_SIZE));
    for (int i = 0; i < HISTOGRAM_SIZE; i++) {
        temp[i] = 0;
    }
    temp[50] = 47;

    int *temp_out =(int*)malloc(sizeof(int)*(HISTOGRAM_SIZE+1));


    CUDA_CHECK( hipMemcpy(image_in_device_serial,temp,HISTOGRAM_SIZE * sizeof(int), hipMemcpyHostToDevice));
    process_image_kernel<<<1,1024 >>>(image_in_device_serial,
            image_out_device_serial);
    CUDA_CHECK( hipMemcpy(temp_out,image_out_device_serial,(1+HISTOGRAM_SIZE) * sizeof(int),hipMemcpyDeviceToHost));
    for (int i = 0; i < HISTOGRAM_SIZE; i++) {
        printf("temp[%d] = %d\n", i,temp_out[i]);
    }
    printf("The minimum is: %d\n", temp_out[HISTOGRAM_SIZE]);
    free(temp);
    free(temp_out);


    //TODO: in a for loop:
    //   1. copy the relevant image from images_in to the GPU memory you allocated
    //   2. invoke GPU kernel on this image
    //   3. copy output from GPU memory to relevant location in images_out_gpu_serial
//    t_finish = get_time_msec(); //Do not change
//    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_serial); // Do not change
//    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not change
//
//    // GPU bulk
//    printf("\n=== GPU Bulk ===\n"); //Do not change
//    //TODO: allocate GPU memory for a all input images and all output images
//    t_start = get_time_msec(); //Do not change
//    //TODO: copy all input images from images_in to the GPU memory you allocated
//    //TODO: invoke a kernel with N_IMAGES threadblocks, each working on a different image
//    //TODO: copy output images from GPU memory to images_out_gpu_bulk
//    t_finish = get_time_msec(); //Do not change
//    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_bulk); // Do not change
//    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not chhange

    return 0;
}
